/*
 * Overdamped Brownian Particle
 *
 * \dot{x} = -V'(x) + a\cos(\omega t) + f + Gaussian, Poissonian and dichotomous noise
 *
 */

#include <stdio.h>
#include <getopt.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define PI 3.14159265358979f

//model
__constant__ float d_amp, d_omega, d_force, d_Dg, d_Dp, d_lambda, d_mean, d_fa, d_fb, d_mua, d_mub;
__constant__ int d_comp;
float h_omega, h_lambda, h_fa, h_fb, h_mua, h_mub, h_mean;
int h_comp;

//simulation
int h_dev, h_block, h_grid, h_spp, h_samples, h_2ndorder, h_initnoise, h_paths, h_periods, h_trans;
long h_threads, h_steps;
__constant__ int d_spp, d_2ndorder, d_samples, d_initnoise, d_paths;

//output
char *h_domain;
char h_domainx, h_domainy;
float h_beginx, h_endx, h_beginy, h_endy;
int h_logx, h_logy, h_points, h_moments, h_traj, h_hist;
__constant__ char d_domainx;
__constant__ int d_moments, d_points;

//vector
float *h_x, *h_fx, *h_xb, *h_w, *h_fw, *h_dx;
float *d_x, *d_fx, *d_w, *d_fw, *d_dx;
int *d_pcd, *d_dcd, *d_dst;
unsigned int *h_seeds, *d_seeds;
hiprandState *d_states;

size_t size_f, size_i, size_ui, size_p;
hiprandGenerator_t gen;

//command line arguments
static struct option options[] = {
    {"amp", required_argument, NULL, 'a'},
    {"omega", required_argument, NULL, 'b'},
    {"force", required_argument, NULL, 'c'},
    {"Dg", required_argument, NULL, 'e'},
    {"Dp", required_argument, NULL, 'f'},
    {"lambda", required_argument, NULL, 'g'},
    {"comp", required_argument, NULL, 'h'},
    {"dev", required_argument, NULL, 'i'},
    {"block", required_argument, NULL, 'j'},
    {"paths", required_argument, NULL, 'k'},
    {"periods", required_argument, NULL, 'l'},
    {"trans", required_argument, NULL, 'm'},
    {"spp", required_argument, NULL, 'n'},
    {"samples", required_argument, NULL, 'o'},
    {"algorithm", required_argument, NULL, 'p'},
    {"mode", required_argument, NULL, 'q'},
    {"domain", required_argument, NULL, 'r'},
    {"domainx", required_argument, NULL, 's'},
    {"domainy", required_argument, NULL, 't'},
    {"logx", required_argument, NULL, 'u'},
    {"logy", required_argument, NULL, 'v'},
    {"points", required_argument, NULL, 'w'},
    {"beginx", required_argument, NULL, 'y'},
    {"endx", required_argument, NULL, 'z'},
    {"beginy", required_argument, NULL, 'A'},
    {"endy", required_argument, NULL, 'B'},
    {"mean", required_argument, NULL, 'C'},
    {"fa", required_argument, NULL, 'D'},
    {"fb", required_argument, NULL, 'E'},
    {"mua", required_argument, NULL, 'F'},
    {"mub", required_argument, NULL, 'G'}
};

void usage(char **argv)
{
    printf("Usage: %s <params> \n\n", argv[0]);
    printf("Model params:\n");
    printf("    -a, --amp=FLOAT         set the harmonic driving amplitude 'a' to FLOAT\n");
    printf("    -b, --omega=FLOAT       set the harmonic driving frequency '\\omega' to FLOAT\n");
    printf("    -c, --force=FLOAT       set the external bias 'F' to FLOAT\n");
    printf("    -e, --Dg=FLOAT          set the Gaussian noise intensity 'D_G' to FLOAT\n");
    printf("    -f, --Dp=FLOAT          set the Poissonian noise intensity 'D_P' to FLOAT\n");
    printf("    -g, --lambda=FLOAT      set the Poissonian kicks frequency '\\lambda' to FLOAT\n");
    printf("    -h, --comp=INT          choose between biased and unbiased Poissonian or dichotomous noise. INT can be one of:\n");
    printf("                            0: biased; 1: unbiased\n");
    printf("    -D, --fa=FLOAT          set the first state of the dichotomous noise 'F_a' to FLOAT\n");
    printf("    -E, --fb=FLOAT          set the second state of the dichotomous noise 'F_b' to FLOAT\n");
    printf("    -F, --mua=FLOAT         set the transition rate of the first state of dichotomous noise '\\mu_a' to FLOAT\n");
    printf("    -G, --mub=FLOAT         set the transition rate of the second state of dichotomous noise '\\mu_b' to FLOAT\n");
    printf("    -C, --mean=FLOAT        if is nonzero, fix the mean value of Poissonian noise or dichotomous noise to FLOAT, matters only for domains p, l, i, j, m or n\n");
    printf("Simulation params:\n");
    printf("    -i, --dev=INT           set the gpu device to INT\n");
    printf("    -j, --block=INT         set the gpu block size to INT\n");
    printf("    -k, --paths=INT        set the number of paths to INT\n");
    printf("    -l, --periods=INT      set the number of periods to INT\n");
    printf("    -m, --trans=FLOAT       specify fraction FLOAT of periods which stands for transients\n");
    printf("    -n, --spp=INT           specify how many integration steps should be calculated\n");
    printf("                            for a single period of the driving force\n");
    printf("    -o, --samples=INT       specify how many integration steps should be calculated for a single kernel call\n");
    printf("    -p, --algorithm=STRING  sets the algorithm. STRING can be one of:\n");
    printf("                            predcorr: simplified weak order 2.0 adapted predictor-corrector\n");
    printf("                            euler: simplified weak order 1.0 regular euler-maruyama\n");
    printf("Output params:\n");
    printf("    -q, --mode=STRING       sets the output mode. STRING can be one of:\n");
    printf("                            moments: the first moment <<v>> and diffusion coefficient\n");
    printf("                            trajectory: ensemble averaged <x>(t) and <x^2>(t)\n");
    printf("                            histogram: the final position x of all paths\n");
    printf("    -r, --domain=STRING     simultaneously scan over one or two model params. STRING can be one of:\n");
    printf("                            1d: only one parameter; 2d: two parameters at once\n");
    printf("    -s, --domainx=CHAR      sets the first domain of the moments. CHAR can be one of:\n");
    printf("                            a: amp; w: omega, f: force; D: Dg; p: Dp; l: lambda; i: fa; j: fb; m: mua; n: mub\n");
    printf("    -t, --domainy=CHAR      sets the second domain of the moments (only if --domain=2d). CHAR can be the same as above.\n");
    printf("    -u, --logx=INT          choose between linear and logarithmic scale of the domainx\n");
    printf("                            0: linear; 1: logarithmic\n");
    printf("    -v, --logy=INT          the same as above but for domainy\n");
    printf("    -w, --points=INT        set the number of samples to generate between begin and end\n");
    printf("    -y, --beginx=FLOAT      set the starting value of the domainx to FLOAT\n");
    printf("    -z, --endx=FLOAT        set the end value of the domainx to FLOAT\n");
    printf("    -A, --beginy=FLOAT      the same as --beginx, but for domainy\n");
    printf("    -B, --endy=FLOAT        the same as --endx, but for domainy\n");
    printf("\n");
}

//parse command line arguments
void parse_cla(int argc, char **argv)
{
    float ftmp;
    int c, itmp;

    while( (c = getopt_long(argc, argv, "a:b:c:e:f:g:h:i:j:k:l:m:n:o:p:q:r:s:t:u:v:w:y:z:A:B:C:D:E:F:G", options, NULL)) != EOF) {
        switch (c) {
            case 'a':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_amp), &ftmp, sizeof(float));
                break;
            case 'b':
                h_omega = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_omega), &h_omega, sizeof(float));
                break;
            case 'c':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_force), &ftmp, sizeof(float));
                break;
            case 'e':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &ftmp, sizeof(float));
                break;
            case 'f':
                ftmp = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &ftmp, sizeof(float));
                break;
            case 'g':
                h_lambda = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &h_lambda, sizeof(float));
                break;
            case 'h':
                h_comp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_comp), &h_comp, sizeof(int));
                break;
            case 'i':
                itmp = atoi(optarg);
                hipSetDevice(itmp);
                break;
            case 'j':
                h_block = atoi(optarg);
                break;
            case 'k':
                h_paths = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_paths), &h_paths, sizeof(int));
                break;
            case 'l':
                h_periods = atoi(optarg);
                break;
            case 'm':
                h_trans = atoi(optarg);
                break;
            case 'n':
                h_spp = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_spp), &h_spp, sizeof(int));
                break;
            case 'o':
                h_samples = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_samples), &h_samples, sizeof(int));
                break;
            case 'p':
                if ( !strcmp(optarg, "predcorr") )
                    h_2ndorder = 1;
                else if ( !strcmp(optarg, "euler") )
                    h_2ndorder = 0;
                hipMemcpyToSymbol(HIP_SYMBOL(d_2ndorder), &h_2ndorder, sizeof(int));
                break;
            case 'q':
                if ( !strcmp(optarg, "moments") ) {
                    h_moments = 1;
                    h_traj = 0;
                    h_hist = 0;
                } else if ( !strcmp(optarg, "trajectory") ) {
                    h_moments = 0;
                    h_traj = 1;
                    h_hist = 0;
                } else if ( !strcmp(optarg, "histogram") ) {
                    h_moments = 0;
                    h_traj = 0;
                    h_hist = 1;
                }
                hipMemcpyToSymbol(HIP_SYMBOL(d_moments), &h_moments, sizeof(int));
                break;
            case 'r':
                h_domain = optarg;
                break;
            case 's':
                h_domainx = optarg[0]; 
                hipMemcpyToSymbol(HIP_SYMBOL(d_domainx), &h_domainx, sizeof(char));
                break;
            case 't':
                h_domainy = optarg[0];
                break;
            case 'u':
                h_logx = atoi(optarg);
                break;
            case 'v':
                h_logy = atoi(optarg);
                break;
            case 'w':
                h_points = atoi(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_points), &h_points, sizeof(int));
                break;
            case 'y':
                h_beginx = atof(optarg);
                break;
            case 'z':
                h_endx = atof(optarg);
                break;
            case 'A':
                h_beginy = atof(optarg);
                break;
            case 'B':
                h_endy = atof(optarg);
                break;
            case 'C':
                h_mean = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mean), &h_mean, sizeof(float));
                break;
            case 'D':
                h_fa = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_fa), &h_fa, sizeof(float));
                break;
            case 'E':
                h_fb = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_fb), &h_fb, sizeof(float));
                break;
            case 'F':
                h_mua = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mua), &h_mua, sizeof(float));
                break;
            case 'G':
                h_mub = atof(optarg);
                hipMemcpyToSymbol(HIP_SYMBOL(d_mub), &h_mub, sizeof(float));
                break;
        }
    }
}

//initialize device random number generator
__global__ void init_dev_rng(unsigned int *d_seeds, hiprandState *d_states)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(d_seeds[idx], 0, 0, &d_states[idx]);
}

__device__ float drift(float l_x, float l_w, float l_amp, float l_force)
{
    return -2.0f*PI*cosf(2.0f*PI*l_x) + l_amp*cosf(l_w) + l_force;
}

__device__ float diffusion(float l_Dg, float l_dt, int l_2ndorder, hiprandState *l_state)
{
    if (l_Dg != 0.0f) {
        float r = hiprand_uniform(l_state);
        float g = sqrtf(2.0f*l_Dg);
        if (l_2ndorder) {
            if ( r <= 1.0f/6 ) {
                return -g*sqrtf(3.0f*l_dt);
            } else if ( r > 1.0f/6 && r <= 2.0f/6 ) {
                return g*sqrtf(3.0f*l_dt);
            } else {
                return 0.0f;
            }
        } else {
            if ( r <= 0.5f ) {
                return -g*sqrtf(l_dt);
            } else {
                return g*sqrtf(l_dt);
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float adapted_jump_poisson(int &npcd, int pcd, float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_lambda != 0.0f) {
        if (pcd <= 0) {
            float ampmean = sqrtf(l_lambda/l_Dp);
           
            npcd = (int) floorf( -logf( hiprand_uniform(l_state) )/l_lambda/l_dt + 0.5f );

            if (l_comp) {
                float comp = sqrtf(l_Dp*l_lambda)*l_dt;
                
                return -logf( hiprand_uniform(l_state) )/ampmean - comp;
            } else {
                return -logf( hiprand_uniform(l_state) )/ampmean;
            }
        } else {
            npcd = pcd - 1;
            if (l_comp) {
                float comp = sqrtf(l_Dp*l_lambda)*l_dt;
                
                return -comp;
            } else {
                return 0.0f;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float adapted_jump_dich(int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt, hiprandState *l_state)
{
    if (l_mua != 0.0f || l_mub != 0.0f) {
        if (dcd <= 0) {
            if (dst == 0) {
                ndst = 1; 
                ndcd = (int) floorf( -logf( hiprand_uniform(l_state) )/l_mub/l_dt + 0.5f );
                return l_fb*l_dt;
            } else {
                ndst = 0;
                ndcd = (int) floorf( -logf( hiprand_uniform(l_state) )/l_mua/l_dt + 0.5f );
                return l_fa*l_dt;
            }
        } else {
            ndcd = dcd - 1;
            if (dst == 0) {
                return l_fa*l_dt;
            } else {
                return l_fb*l_dt;
            }
        }
    } else {
        return 0.0f;
    }
}

__device__ float regular_jump_poisson(float l_lambda, float l_Dp, int l_comp, float l_dt, hiprandState *l_state)
{
    if (l_lambda != 0.0f) {
        float mu, ampmean, comp, s;
        int i;
        unsigned int n;

        mu = l_lambda*l_dt;
        ampmean = sqrtf(l_lambda/l_Dp);
        comp = sqrtf(l_Dp*l_lambda)*l_dt;
        n = hiprand_poisson(l_state, mu);
        s = 0.0f;
            for (i = 0; i < n; i++) {
                s += -logf( hiprand_uniform(l_state) )/ampmean;
            }
        if (l_comp) s -= comp;
        return s;
    } else {
        return 0.0f;
    }
}

/* simplified weak order 2.0 adapted predictor-corrector scheme
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 503, p. 532 )
*/
__device__ void predcorr(float &corrl_x, float l_x, float &corrl_w, float l_w, int &npcd, int pcd, hiprandState *l_state, \
                         float l_amp, float l_omega, float l_force, float l_Dg, int l_2ndorder, float l_Dp, float l_lambda, int l_comp, \
                         int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt)
{
    float l_xt, l_xtt, l_wt, l_wtt, predl_x, predl_w;

    l_xt = drift(l_x, l_w, l_amp, l_force);
    l_wt = l_omega;

    predl_x = l_x + l_xt*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state);
    predl_w = l_w + l_wt*l_dt;

    l_xtt = drift(predl_x, predl_w, l_amp, l_force);
    l_wtt = l_omega;

    predl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state);
    predl_w = l_w + 0.5f*(l_wt + l_wtt)*l_dt;

    l_xtt = drift(predl_x, predl_w, l_amp, l_force);
    l_wtt = l_omega;

    corrl_x = l_x + 0.5f*(l_xt + l_xtt)*l_dt + diffusion(l_Dg, l_dt, l_2ndorder, l_state) + adapted_jump_poisson(npcd, pcd, l_lambda, l_Dp, l_comp, l_dt, l_state) + adapted_jump_dich(ndcd, dcd, ndst, dst, l_fa, l_fb, l_mua, l_mub, l_dt, l_state);
    corrl_w = l_w + 0.5f*(l_wt + l_wtt)*l_dt;
}

/* simplified weak order 1.0 regular euler-maruyama scheme 
( see E. Platen, N. Bruti-Liberati; Numerical Solution of Stochastic Differential Equations with Jumps in Finance; Springer 2010; p. 508, 
  C. Kim, E. Lee, P. Talkner, and P.Hanggi; Phys. Rev. E 76; 011109; 2007 ) 
*/
__device__ void eulermaruyama(float &nl_x, float l_x, float &nl_w, float l_w, hiprandState *l_state, \
                         float l_amp, float l_omega, float l_force, float l_Dg, int l_2ndorder, float l_Dp, float l_lambda, int l_comp, \
                         int &ndcd, int dcd, int &ndst, int dst, float l_fa, float l_fb, float l_mua, float l_mub, float l_dt)
{
    float l_xt, l_wt;

    l_xt = l_x + drift(l_x, l_w, l_amp, l_force)*l_dt
               + diffusion(l_Dg, l_dt, l_2ndorder, l_state)
               + regular_jump_poisson(l_lambda, l_Dp, l_comp, l_dt, l_state)
               + adapted_jump_dich(ndcd, dcd, ndst, dst, l_fa, l_fb, l_mua, l_mub, l_dt, l_state);
    l_wt = l_w + l_omega*l_dt;

    nl_x = l_xt;
    nl_w = l_wt;
}

//reduce periodic variable to the base domain
__global__ void fold(float *d_x, float *d_fx, float p)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    float l_x, l_fx, f;

    l_x = d_x[idx];
    l_fx = d_fx[idx];
    
    if (fabsf(l_x) >= p) {
        f = floorf(l_x/p)*p;
        l_x = l_x - f;
        l_fx = l_fx + f;
    }

    d_x[idx] = l_x;
    d_fx[idx] = l_fx;
}

//unfold periodic variable
void unfold(float *x, float *fx)
{
    long i;

    for (i = 0; i < h_threads; i++) {
        x[i] = x[i] + fx[i];
    }
}

//actual simulation kernel
__global__ void run_sim(float *d_x, float *d_w, float *d_dx, int *d_pcd, int *d_dcd, int *d_dst, hiprandState *d_states)
{
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    //cache path and model parameters in local variables
    float l_x, l_w, l_dx; 
    hiprandState l_state;

    l_x = d_x[idx];
    l_w = d_w[idx];
    l_state = d_states[idx];

    float l_amp, l_omega, l_force, l_Dg, l_Dp, l_lambda, l_mean, l_fa, l_fb, l_mua, l_mub;
    int l_comp, l_2ndorder;
    int l_moments;

    l_amp = d_amp;
    l_omega = d_omega;
    l_force = d_force;
    l_Dg = d_Dg;
    l_Dp = d_Dp;
    l_lambda = d_lambda;
    l_mean = d_mean;
    l_comp = d_comp;
    l_fa = d_fa;
    l_fb = d_fb;
    l_mua = d_mua;
    l_mub = d_mub;
    l_2ndorder = d_2ndorder;
    l_moments = d_moments;
   
    //run simulation for multiple values of the system parameters
    if (l_moments) {
        long ridx = (idx/d_paths) % d_points;
        l_dx = d_dx[ridx];

        switch(d_domainx) {
            case 'a':
                l_amp = l_dx;
                break;
            case 'w':
                l_omega = l_dx;
                break;
            case 'f':
                l_force = l_dx;
                break;
            case 'D':
                l_Dg = l_dx;
                break;
            case 'p':
                l_Dp = l_dx;
                if (l_mean != 0.0f) l_lambda = (l_mean*l_mean)/l_Dp;
                break;
            case 'l':
                l_lambda = l_dx;
                if (l_mean != 0.0f) l_Dp = (l_mean*l_mean)/l_lambda;
                break;
            case 'i':
                l_fa = l_dx;
                if (l_comp == 1) {
                    l_mua = -l_fa*l_mub/l_fb;
                    //l_fb = -l_fa*l_mub/l_mua;
                } else if (l_mean != 0.0f) {
                    l_mua = (l_fa - l_mean)*l_mub/(l_mean - l_fb);
                    //l_fb = (l_mean*(l_mua + l_mub) - l_fa*l_mub)/l_mua;
                }
                break;
            case 'j':
                l_fb = l_dx;
                if (l_comp == 1) {
                    l_mub = -l_fb*l_mua/l_fa;
                    //l_fa = -l_fb*l_mua/l_mub;
                } else if (l_mean != 0.0f) {
                    l_mub = (l_fb - l_mean)*l_mua/(l_mean - l_fa);
                    //l_fa = (l_mean*(l_mua + l_mub) - l_fb*l_mua)/l_mub;
                }
                break;
            case 'm':
                l_mua = l_dx;
                if (l_comp == 1) {
                    l_fa = -l_fb*l_mua/l_mub;
                    //l_mub = -l_fb*l_mua/l_fa;
                } else if (l_mean != 0.0f) {
                    l_fa = (l_mean*(l_mua + l_mub) - l_fb*l_mua)/l_mub;
                    //l_mub = (l_fb - l_mean)*l_mua/(l_mean - l_fa);
                }
                break;
            case 'n':
                l_mub = l_dx;
                if (l_comp == 1) {
                    l_fb = -l_fa*l_mub/l_mua;
                    //l_mua = -l_fa*l_mub/l_fb;
                } else if (l_mean != 0.0f) {
                    l_fb = (l_mean*(l_mua + l_mub) - l_fa*l_mub)/l_mua;
                    //l_mua = (l_fa - l_mean)*l_mub/(l_mean - l_fb);
                }
                break;
        }
    }

    //step size
    float l_dt, tmp;

    l_dt = 2.0f*PI/l_omega;

    if (l_lambda != 0.0f) {
        if (l_2ndorder) {
            tmp = 1.0f/l_lambda;
            if (tmp < l_dt) l_dt = tmp;
        }
    }

    if (l_mua != 0.0f || l_mub != 0.0f) {
        float taua, taub;

        taua = 1.0f/l_mua;
        taub = 1.0f/l_mub;
        
        if (taua < taub) {
            tmp = taua;
        } else {
            tmp = taub;
        }

        if (tmp < l_dt) l_dt = tmp;
    }

    int l_spp;

    l_spp = d_spp;
    l_dt /= l_spp;

    //number of steps
    int l_samples;

    l_samples = d_samples;

    //jump countdowns
    int l_initnoise, l_pcd, l_dcd, l_dst;

    l_initnoise = d_initnoise;

    if (l_initnoise) {

        if (l_lambda != 0.0f) {
            if (l_2ndorder) {
                l_pcd = (int) floorf( -logf( hiprand_uniform(&l_state) )/l_lambda/l_dt + 0.5f );
            }
        }

        if (l_mua != 0.0f || l_mub != 0.0f) {
            float rn;
            rn = hiprand_uniform(&l_state);

            if (rn < 0.5f) {
                l_dst = 0;
                l_dcd = (int) floorf( -logf( hiprand_uniform(&l_state) )/l_mua/l_dt + 0.5f);
            } else {
                l_dst = 1;
                l_dcd = (int) floorf( -logf( hiprand_uniform(&l_state) )/l_mub/l_dt + 0.5f);
            }
        }

    } else {
        
        if (l_lambda != 0.0f) {
            if (l_2ndorder) {
                l_pcd = d_pcd[idx];
            }
        }
    
        if (l_mua != 0.0f || l_mub != 0.0f) {
            l_dcd = d_dcd[idx];
            l_dst = d_dst[idx];
        }

        int i;
    
        for (i = 0; i < l_samples; i++) {
            //algorithm
            if (l_2ndorder) {
                predcorr(l_x, l_x, l_w, l_w, l_pcd, l_pcd, &l_state, l_amp, l_omega, l_force, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, \
                         l_dcd, l_dcd, l_dst, l_dst, l_fa, l_fb, l_mua, l_mub, l_dt);
            } else {
                eulermaruyama(l_x, l_x, l_w, l_w, &l_state, l_amp, l_omega, l_force, l_Dg, l_2ndorder, l_Dp, l_lambda, l_comp, \
                         l_dcd, l_dcd, l_dst, l_dst, l_fa, l_fb, l_mua, l_mub, l_dt);
            }
        }
    }

    //write back path parameters to the global memory
    d_x[idx] = l_x;
    d_w[idx] = l_w;
    d_pcd[idx] = l_pcd;
    d_dcd[idx] = l_dcd;
    d_dst[idx] = l_dst;
    d_states[idx] = l_state;
}

//prepare simulation
void prepare()
{
    //grid size
    h_paths = (h_paths/h_block)*h_block;
    h_threads = h_paths;

    if (h_moments) h_threads *= h_points;

    h_grid = h_threads/h_block;

    //number of steps
    h_steps = h_periods*h_spp;
     
    //host memory allocation
    size_f = h_threads*sizeof(float);
    size_i = h_threads*sizeof(int);
    size_ui = h_threads*sizeof(unsigned int);
    size_p = h_points*sizeof(float);

    h_x = (float*)malloc(size_f);
    h_fx = (float*)malloc(size_f);
    h_w = (float*)malloc(size_f);
    h_fw = (float*)malloc(size_f);
    h_seeds = (unsigned int*)malloc(size_ui);

    //create & initialize host rng
    hiprandCreateGeneratorHost(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    hiprandGenerate(gen, h_seeds, h_threads);
 
    //device memory allocation
    hipMalloc((void**)&d_x, size_f);
    hipMalloc((void**)&d_fx, size_f);
    hipMalloc((void**)&d_w, size_f);
    hipMalloc((void**)&d_fw, size_f);
    hipMalloc((void**)&d_pcd, size_i);
    hipMalloc((void**)&d_dcd, size_i);
    hipMalloc((void**)&d_dst, size_i);
    hipMalloc((void**)&d_seeds, size_ui);
    hipMalloc((void**)&d_states, h_threads*sizeof(hiprandState));

    //copy seeds from host to device
    hipMemcpy(d_seeds, h_seeds, size_ui, hipMemcpyHostToDevice);

    //initialization of device rng
    init_dev_rng<<<h_grid, h_block>>>(d_seeds, d_states);

    free(h_seeds);
    hipFree(d_seeds);

    //moments specific requirements
    h_xb = (float*)malloc(size_f);
    h_dx = (float*)malloc(size_p);

    float dxtmp = h_beginx;
    float dxstep = (h_endx - h_beginx)/h_points;

    int i;
        
    //set domainx
    for (i = 0; i < h_points; i++) {
        if (h_logx) {
            h_dx[i] = exp10f(dxtmp);
        } else {
            h_dx[i] = dxtmp;
        }
        dxtmp += dxstep;
    }
        
    hipMalloc((void**)&d_dx, size_p);
    
    hipMemcpy(d_dx, h_dx, size_p, hipMemcpyHostToDevice);
}

void copy_to_dev()
{
    hipMemcpy(d_x, h_x, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_fx, h_fx, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_f, hipMemcpyHostToDevice);
    hipMemcpy(d_fw, h_fw, size_f, hipMemcpyHostToDevice);
}

void copy_from_dev()
{
    hipMemcpy(h_x, d_x, size_f, hipMemcpyDeviceToHost);
    hipMemcpy(h_fx, d_fx, size_f, hipMemcpyDeviceToHost);
}

//set initial conditions
void initial_conditions()
{
    hiprandGenerateUniform(gen, h_x, h_threads); //x in (0,1]
    hiprandGenerateUniform(gen, h_w, h_threads);

    long i;

    for (i = 0; i < h_threads; i++) {
        h_w[i] *= 2.0f*PI; //w in (0,2\pi]
    }

    memset(h_fx, 0.0f, size_f);
    memset(h_fw, 0.0f, size_f);

    copy_to_dev();
}

//calculate the first moment of <v> and diffusion coefficient
void moments(float *av, float *dc)
{
    float sx, sx2, sxb, dt, tempo, tmp, taua, taub;
    int i, j;

    copy_from_dev();

    unfold(h_x, h_fx);

    for (j = 0; j < h_points; j++) {
        sx = 0.0f;
        sx2 = 0.0f;
        sxb = 0.0f;

        for (i = 0; i < h_paths; i++) {
            sx += h_x[j*h_paths + i];
            sx2 += h_x[j*h_paths + i]*h_x[j*h_paths + i];
            sxb += h_xb[j*h_paths + i];
        }

        //external driving
        if (h_domainx == 'w') {
            tempo = 2.0f*PI/h_dx[j];
        } else {
            tempo = 2.0f*PI/h_omega;
        }
       
        dt = tempo;

        //Poissonian
        if (h_lambda != 0.0f && h_2ndorder) {
            if (h_domainx == 'l') {
                tmp = 1.0f/h_dx[j];
            } else if (h_domainx == 'p' && h_mean != 0.0f) {
                tmp = 1.0f/(h_mean*h_mean/h_dx[j]);
            } else {
                tmp = 1.0f/h_lambda;
            }

            if (tmp < tempo) dt = tmp;
        }

        //Dichotomous
        if (h_mua != 0.0f || h_mub != 0.0f) {
            if (h_domainx == 'm') {
                taua = 1.0f/h_dx[j];
                taub = 1.0f/h_mub;

                /*if (h_comp) {
                    tmp = 1.0f/(-h_fb*h_dx[j]/h_fa);
                } else if (h_mean != 0.0f) {
                    tmp = 1.0f/((h_fb - h_mean)*h_dx[j]/(h_mean - h_fa));
                } else {*/
                    tmp = taub;
                //}
            
                if (taua <= tmp) {
                    if (taua < tempo) dt = taua;
                } else {
                    if (tmp < tempo) dt = tmp;
                }
            } else if (h_domainx == 'n') {
                taua = 1.0f/h_mua;
                taub = 1.0f/h_dx[j];

                /*if (h_comp) {
                    tmp = 1.0f/(-h_fa*h_dx[j]/h_fb);
                } else if (h_mean != 0.0f) {
                    tmp = 1.0f/((h_fa - h_mean)*h_dx[j]/(h_mean - h_fb));
                } else {*/
                    tmp = taua;
                //}

                if (taub <= tmp) {
                    if (taub < tempo) dt = taub;
                } else {
                    if (tmp < tempo) dt = tmp;
                }
            } else if (h_domainx == 'i') {
                taua = 1.0f/h_mua;
                taub = 1.0f/h_mub;

                if (h_comp) {
                    tmp = 1.0f/(-h_dx[j]*h_mub/h_fb);
                } else if (h_mean != 0.0f) {
                    tmp = 1.0f/((h_dx[j] - h_mean)*h_mub/(h_mean - h_fb));
                } else {
                    tmp = taua;
                }

                if (taub <= tmp) {
                    if (taub < tempo) dt = taub;
                } else {
                    if (tmp < tempo) dt = tmp;
                }
            } else if (h_domainx == 'j') {
                taua = 1.0f/h_mua;
                taub = 1.0f/h_mub;

                if (h_comp) {
                    tmp = 1.0f/(-h_dx[j]*h_mua/h_fa);
                } else if (h_mean != 0.0f) {
                    tmp = 1.0f/((h_dx[j] - h_mean)*h_mua/(h_mean - h_fa));
                } else {
                    tmp = taub;
                }

                if (taua <= tmp) {
                    if (taua < tempo) dt = taua;
                } else {
                    if (tmp < tempo) dt = tmp;
                }
            } else {
                taua = 1.0f/h_mua;
                taub = 1.0f/h_mub;

                if (taua < taub) {
                    if (taua < tempo) dt = taua;
                } else {
                    if (taub < tempo) dt = taub;
                }
            }
        }

        dt /= h_spp;

        sx /= h_paths;
        sx2 /= h_paths;
        sxb /= h_paths;
        av[j] = (sx - sxb)/( (h_periods - h_trans)*h_spp*dt );
        dc[j] = (sx2 - sx*sx)/(2.0f*h_steps*dt);
    }
}

//calculate ensemble average
void ensemble_average(float *h_x, float &sx, float &sx2)
{
    long i;

    sx = 0.0f;
    sx2 = 0.0f;

    for (i = 0; i < h_threads; i++) {
        sx += h_x[i];
        sx2 += h_x[i]*h_x[i];
    }

    sx /= h_threads;
    sx2 /= h_threads;
}

//free memory
void finish()
{
    free(h_x);
    free(h_fx);
    free(h_xb);
    free(h_w);
    free(h_fw);
    
    hiprandDestroyGenerator(gen);
    hipFree(d_x);
    hipFree(d_fx);
    hipFree(d_w);
    hipFree(d_fw);
    hipFree(d_pcd);
    hipFree(d_dcd);
    hipFree(d_dst);
    hipFree(d_states);
    
    free(h_xb);
    free(h_dx);

    hipFree(d_dx);
}

int main(int argc, char **argv)
{
    parse_cla(argc, argv);
    if (!h_moments && !h_traj && !h_hist) {
        usage(argv);
        return -1;
    }

    prepare();

    initial_conditions();

    h_initnoise = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));

    if ( (h_lambda != 0.0f && h_2ndorder) || (h_mua != 0.0f || h_mub != 0.0f) ) {
        h_initnoise = 1;
        hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));

        run_sim<<<h_grid, h_block>>>(d_x, d_w, d_dx, d_pcd, d_dcd, d_dst, d_states);

        h_initnoise = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));
    }

    //asymptotic long time average velocity <<v>>, <<v^2>> and diffusion coefficient
    if (h_moments) {
        float *av, *dc;
        long i;

        av = (float*)malloc(size_p);
        dc = (float*)malloc(size_p);

        if ( !strcmp(h_domain, "1d") ) { 

            for (i = 0; i < h_steps; i += h_samples) {
                run_sim<<<h_grid, h_block>>>(d_x, d_w, d_dx, d_pcd, d_dcd, d_dst, d_states);
                fold<<<h_grid, h_block>>>(d_x, d_fx, 1.0f);
                fold<<<h_grid, h_block>>>(d_w, d_fw, (2.0f*PI));
                if ( i == h_trans*h_spp) {
                    hipMemcpy(h_xb, d_x, size_f, hipMemcpyDeviceToHost);
                    hipMemcpy(h_fx, d_fx, size_f, hipMemcpyDeviceToHost);
                    unfold(h_xb, h_fx);
                }
            }

            moments(av, dc);

            printf("#%c <<v>> D_x\n", h_domainx);
            for (i = 0; i < h_points; i++) {
                printf("%e %e %e\n", h_dx[i], av[i], dc[i]);
            }

        } else {
            float h_dy, dytmp, dystep;
            int j, k;
            
            dytmp = h_beginy;
            dystep = (h_endy - h_beginy)/h_points;
            
            printf("#%c %c <<v>> D_x\n", h_domainx, h_domainy);
            
            for (k = 0; k < h_points; k++) {
                if (h_logy) {
                    h_dy = exp10f(dytmp);
                } else {
                    h_dy = dytmp;
                }

                switch(h_domainy) {
                    case 'a':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_amp), &h_dy, sizeof(float));
                        break;
                    case 'w':
                        h_omega = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_omega), &h_omega, sizeof(float));
                        break;
                    case 'f':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_force), &h_dy, sizeof(float));
                        break;
                    case 'D':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_Dg), &h_dy, sizeof(float));
                        break;
                    case 'p':
                        hipMemcpyToSymbol(HIP_SYMBOL(d_Dp), &h_dy, sizeof(float));
                        break;
                    case 'l':
                        h_lambda = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_lambda), &h_lambda, sizeof(float));
                        break;
                    case 'i':
                        h_fa = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_fa), &h_fa, sizeof(float));
                        break;
                    case 'j':
                        h_fb = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_fb), &h_fb, sizeof(float));
                        break;
                    case 'm':
                        h_mua = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_mua), &h_mua, sizeof(float));
                        break;
                    case 'n':
                        h_mub = h_dy;
                        hipMemcpyToSymbol(HIP_SYMBOL(d_mub), &h_mub, sizeof(float));
                        break;
                }
 
                for (i = 0; i < h_steps; i += h_samples) {
                    run_sim<<<h_grid, h_block>>>(d_x, d_w, d_dx, d_pcd, d_dcd, d_dst, d_states);
                    fold<<<h_grid, h_block>>>(d_x, d_fx, 1.0f);
                    fold<<<h_grid, h_block>>>(d_w, d_fw, (2.0f*PI));
                    if ( i == h_trans*h_spp) {
                        hipMemcpy(h_xb, d_x, size_f, hipMemcpyDeviceToHost);
                        hipMemcpy(h_fx, d_fx, size_f, hipMemcpyDeviceToHost);
                        unfold(h_xb, h_fx);
                    }
                }
 
                moments(av, dc);
                
                for (j = 0; j < h_points; j++) {
                    printf("%e %e %e %e\n", h_dx[j], h_dy, av[j], dc[j]);
                }

                //blank line for plotting purposes
                printf("\n");

                initial_conditions();

                if ( (h_lambda != 0.0f && h_2ndorder) || (h_mua != 0.0f || h_mub != 0.0f) ) {
                    h_initnoise = 1;
                    hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));

                    run_sim<<<h_grid, h_block>>>(d_x, d_w, d_dx, d_pcd, d_dcd, d_dst, d_states);

                    h_initnoise = 0;
                    hipMemcpyToSymbol(HIP_SYMBOL(d_initnoise), &h_initnoise, sizeof(int));
                }

                dytmp += dystep;
           }
        }

        free(av);
        free(dc);
    }

    //ensemble averaged trajectory <x>(t) and <x^2>(t)
    if (h_traj) {
        float t, sx, sx2, dt, tmp, taua, taub;
        long i;

        dt = 2.0f*PI/h_omega;
        tmp = dt;

        if (h_lambda != 0.0f && h_2ndorder) tmp = 1.0f/h_lambda;

        if (h_mua != 0.0f || h_mub != 0.0f) {
            taua = 1.0f/h_mua;
            taub = 1.0f/h_mub;

            if (taua < taub) {
                tmp = taua;
            } else {
                tmp = taub;
            }
        }

        if (tmp < dt) dt = tmp;

        dt /= h_spp;

        printf("#t <x> <x^2>\n");
        
        for (i = 0; i < h_steps; i += h_samples) {
            run_sim<<<h_grid, h_block>>>(d_x, d_w, d_dx, d_pcd, d_dcd, d_dst, d_states);
            copy_from_dev();
            unfold(h_x, h_fx);
            t = i*dt;
            ensemble_average(h_x, sx, sx2);
            printf("%e %e %e\n", t, sx, sx2);
            fold<<<h_grid, h_block>>>(d_x, d_fx, 1.0f);
            fold<<<h_grid, h_block>>>(d_w, d_fw, (2.0f*PI));
        }
    }

    //the final position x of all paths
    if (h_hist) {
        long i;

        for (i = 0; i < h_steps; i += h_samples) {
            run_sim<<<h_grid, h_block>>>(d_x, d_w, d_dx, d_pcd, d_dcd, d_dst, d_states);
            fold<<<h_grid, h_block>>>(d_x, d_fx, 1.0f);
            fold<<<h_grid, h_block>>>(d_w, d_fw, (2.0f*PI));
        }
        
        copy_from_dev();

        printf("#x\n");
        
        for (i = 0; i < h_threads; i++) {
            printf("%e\n", h_x[i]); 
        }
    }

    finish();

    return 0;
}
